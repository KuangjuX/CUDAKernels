#include "hip/hip_runtime.h"
#include "kernels/warp_reduce.hpp"
#include "warp/mod.hpp"

namespace cudakernels::kernels {
template <typename Element>
__global__ void reduce_sum_kernel(const Element* input, Element* output,
                                  int size, int thread_size) {
    // TODO: WARP_SIZE should be a template parameter
    constexpr int WARP_SIZE = 32;
    constexpr int THREADS_NUM = thread_size;
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int lane_id = tid % WARP_SIZE;
    int warp_id = tid / WARP_SIZE;

    // always <= 32 warps per block(limited by 1024 threads per block)
    constexpr int WARP_NUM = (THREADS_NUM + WARP_SIZE - 1) / WARP_SIZE;

    __shared__ Element shared[WARP_NUM];

    // 得到当前线程应该处理的数据
    Element sum = (idx < size) ? input[idx] : 0;

    // 使用 warp 做累加求和
    warp::warp_reduce_sum<Element, WARP_SIZE>(sum);

    // 取 lane_id 为 0 的结果为该 warp 处理的结果
    if (lane_id == 0) {
        shared[warp_id] = sum;
    }

    __syncthreads();

    // WARP_NUM 应该小于 32，因此可以使用一个 warp 来计算所有 warps 的结果
    sum = (lane_id < WARP_NUM) ? shared[lane_id] : 0;

    // 使用一个 warp 对所有 warps 计算出的结果做累加
    if (warp_id == 0) {
        warp::warp_reduce_sum<Element, WARP_NUM>(sum);
    }

    // 对所有 blocks 进行累加得到 reduce 的结果
    if (tid == 0) atomicAdd(output, sum);
}

template <typename Element>
__global__ void reduce_max_kernel(const Element* input, Element* output,
                                  int size, int thread_size) {
    constexpr int WARP_SIZE = 32;
    constexpr int THREADS_NUM = thread_size;
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int lane_id = tid % WARP_SIZE;
    int warp_id = tid / WARP_SIZE;

    constexpr int WARP_NUM = (THREADS_NUM + WARP_SIZE - 1) / WARP_SIZE;

    __shared__ Element shared[WARP_NUM];

    // 得到当前线程应该处理的数据
    Element max = (idx < size) ? input[idx] : 0;

    // 使用 warp 做 reduce 求最大值
    warp::warp_reduce_sum<Element, WARP_SIZE>(max);

    // 取 lane_id 为 0 的结果为该 warp 处理的结果
    if (lane_id == 0) {
        shared[warp_id] = max;
    }

    __syncthreads();

    // WARP_NUM 应该小于 32，因此可以使用一个 warp 来计算所有 warps 的结果
    max = (lane < WARP_NUM) ? shared[lane] : 0;

    // 使用一个 warp 对所有 warps 计算出的结果做 reduce max
    if (warp_id == 0) {
        warp::warp_reduce_max<Element, WARP_NUM>(max);
    }

    // 对所有 blocks 进行 reduce max 得到结果
    if (tid == 0) atomicMax(output, max);
}
}  // namespace cudakernels::kernels