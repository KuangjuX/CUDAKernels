#include "hip/hip_runtime.h"
#include "kernels/flash_attn/flash_attn_f32.hpp"

#include <stdio.h>

namespace cuda_kernels::kernels {
/**
 * @brief Flash Attention forward kernel
 * @param[in] Q Query tensor
 * @param[in] K Key tensor
 * @param[in] V Value tensor
 * @param[in] N Batch size
 * @param[in] d Multi-head dimension
 * @param[in] Tc Sequence length
 * @param[in] Tr Sequence length
 * @param[in] Bc Block size
 * @param[in] Br Block size
 * @param[in] softmax_scale Softmax scale
 * @param[out] l softmax sum tensor
 * @param[out] m softmax max tensor
 * @param[out] O Output tensor
 */
__global__ void flash_attn_fwd_f32_kernel(
    const float* Q, const float* K, const float* V, const int N, const int d,
    const int Tc, const int Tr, const int Bc, const int Br,
    const float softmax_scale, float* l, float* m, float* O) {
    int tid = threadIdx.x;
    // Batch, head 在 thread blocks 上进行并行
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // 一个 thread block 处理一个 softmax(QK^T)V

    // 获得 Q, K, V 的偏移
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);
    // 获得 l, m 的偏移
    int lm_offset = (bx * gridDim.y * N) + (by * N);

    // 为 Q，K，V，S 定义 SRAM
    extern __shared__ float sram[];

    int tile_size = Bc * d;
    float* Qi = sram;
    float* Kj = Qi + tile_size;
    float* Vj = Kj + tile_size;
    float* S = Vj + tile_size;

    // 外层循环将 K, V 进行 tile 并加载到 SRAM 中
    for (int j = 0; j < Tc; ++j) {
        // 加载 Kj, Vj 到 SRAM
        for (int x = 0; x < d; ++x) {
            // 看起来没有进行内存合并访问？
            Kj[(tid * d) + x] = K[qkv_offset + (tile_size * j) + (tid * d) + x];
            Vj[(tid * d) + x] = V[qkv_offset + (tile_size * j) + (tid * d) + x];
        }

        // for (int x = 0; x < d; ++x) {
        //     // 打印 Kj, Vj
        //     printf("Kj[%d]: %f, Vj[%d]: %f\n", (tid * d) + x, Kj[(tid * d) +
        //     x],
        //            (tid * d) + x, Vj[(tid * d) + x]);
        // }

        // 同步所有线程，内层循环可以正确使用 Kj, Vj
        __syncthreads();

        // 内层循环
        for (int i = 0; i < Tr; ++i) {
            // 加载 Qi 到 SRAM，l，m 到寄存器
            for (int x = 0; x < d; ++x) {
                // 每个线程处理一行(一个 d)
                Qi[(tid * d) + i] =
                    Q[qkv_offset + (tile_size * i) + (tid * d) + i];
            }

            // for (int x = 0; x < d; ++x) {
            //     // 打印 Qi
            //     printf("Qi[%d]: %f\n", (tid * d) + i, Qi[(tid * d) + i]);
            // }
            // 一次循环加载一次 l，m
            float row_m_prev = m[lm_offset + (Br * i) + tid];
            float row_l_prev = l[lm_offset + (Br * i) + tid];

            // S = Qk^T, row_m = rowmax(S)
            float row_m = -INFINITY;
            // 循环一个切块的大小
            for (int y = 0; y < Bc; ++y) {
                float sum = 0;
                // S = Qk^T
                // 一个线程处理一个 d，每次都和 K 进行计算
                for (int x = 0; x < d; ++x) {
                    sum += Qi[(tid * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                S[(Bc * tid) + y] = sum;

                if (sum > row_m) row_m = sum;
            }

            // P = exp(S - row_m), row_l = rowsum(P)
            float row_l = 0;
            for (int y = 0; y < Bc; ++y) {
                S[(Bc * tid) + y] = __expf(S[(Bc * tid) + y] - row_m);
                row_l += S[(Bc * tid) + y];
            }

            // 打印 S
            // for (int y = 0; y < Bc; ++y) {
            //     printf("S[%d]: %f\n", (Bc * tid) + y, S[(Bc * tid) + y]);
            // }

            // 计算新的 m 和 l
            float row_m_new = max(row_m, row_m_prev);
            // online softmax 的计算
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) +
                              (__expf(row_m - row_m_new) * row_l);

            // printf("row_m_new: %f, row_l_new: %f\n", row_m_new, row_l_new);

            // 将 O,l,m 写回到 HBM
            for (int x = 0; x < d; ++x) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; ++y) {
                    pv += S[(Bc * tid) + y] * Vj[(y * d) + x];
                }
                // 同样是基于迭代式对 O 进行更新
                O[qkv_offset + (tile_size * i) + (tid * d) + x] =
                    (1 / row_l_new) *
                        ((row_l_prev * __expf(row_m_prev - row_m_new))) *
                        O[qkv_offset + (tile_size * i) + (tid * d) + x] +
                    (__expf(row_m - row_m_new) * pv);
            }

            // 打印 O
            // for (int x = 0; x < d; ++x) {
            //      printf("O[%d]: %f\n", (tid * d) + x,
            //           O[qkv_offset + (tile_size * i) + (tid * d) + x]);
            // }

            m[lm_offset + (Br * i) + tid] = row_m_new;
            l[lm_offset + (Br * i) + tid] = row_l_new;
        }
        // 同步内层循环
        __syncthreads();
    }
}

void flash_attn_fwd(const torch::Tensor& Q, const torch::Tensor& K,
                    const torch::Tensor& V, torch::Tensor& O) {
    // 设置块大小，其中 Bc = ceil(M/4d), Br = min(ceil(M/4d), d)
    // 这里全部使用 32 作为块大小
    const int Bc = 32;
    const int Br = 32;

    const int B = Q.size(0);
    const int nh = Q.size(1);
    const int N = Q.size(2);
    const int d = Q.size(3);

    printf("B: %d, nh: %d, N: %d, d: %d\n", B, nh, N, d);

    const int Tc = ceil((float)N / Bc);
    const int Tr = ceil((float)N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    auto type = Q.dtype();

    // auto O = torch::zeros_like(Q);
    auto l = torch::zeros({B, nh, N}, torch::kFloat32);
    auto m = torch::full({B, nh, N}, -FP_INFINITE, torch::kFloat32);
    torch::Device device(torch::kCUDA);

    l = l.to(device);
    m = m.to(device);

    // 计算 SRAM 的大小
    const int sram_size =
        (3 * Bc * d * sizeof(float)) + (Bc * Br * sizeof(float));

    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock,
                           0);
    printf("Max shared memory per block: %d, requested shared memory: %d\n",
           max_sram_size, sram_size);

    dim3 grid_dim(B, nh);
    dim3 block_dim(Bc);

    if (type == torch::kFloat32) {
        flash_attn_fwd_f32_kernel<<<grid_dim, block_dim, sram_size>>>(
            Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(), N, d,
            Tc, Tr, Bc, Br, softmax_scale, l.data_ptr<float>(),
            m.data_ptr<float>(), O.data_ptr<float>());
    } else {
        throw std::runtime_error("Unsupported data type");
    }
}
}  // namespace cuda_kernels::kernels